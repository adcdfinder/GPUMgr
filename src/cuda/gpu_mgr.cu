#ifndef GPU_MGR_CU__
#define GPU_MGR_CU__

#include <mutex>
#include <condition_variable>
#include <algorithm>
#include <iostream>
#include <unistd.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "gpu_mgr/gpu_mgr.cuh"
#include "gpu_utils.hpp"

struct HostFuncData
{
  bool IsNotify;
  std::function<void(void)> *Post_Function;
  std::mutex *gopt_cv_mutex;
  std::condition_variable *gopt_cv;
  hipStream_t *gopt_stm;
  void *gpu_opt;
};

hipStream_t *m_stream;                // Only initialized by GPU_Init at gpu_mgr.cu
hipStream_t *m_streamWithlow;
hipStream_t *m_streamWithhigh;
std::function<void (std::mutex*,std::condition_variable*)> notifyGpuMgr; // Only assigned by GPU_Init at gpu_mgr.cu

void GPU_Init(int policy_type,int stream_num,std::function<void (std::mutex*,std::condition_variable*)> notify_callback)
{
  // Create CUDA stream array dynamically
  m_stream = (hipStream_t *)malloc(stream_num * sizeof(hipStream_t *));
  m_streamWithlow = (hipStream_t *)malloc(stream_num * sizeof(hipStream_t *));
  m_streamWithhigh = (hipStream_t *)malloc(sizeof(hipStream_t *));

  // Create CUDA streams no priority
  for (int i = 0; i < stream_num; i++)
    hipStreamCreate(&(m_stream[i]));

  // get the range of stream priorities for this device
  int priority_high, priority_low;
  hipDeviceGetStreamPriorityRange(&priority_low,&priority_high);
  // create streams with highest and lowest available priorities
  hipStream_t st_high, st_low;
  hipStreamCreateWithPriority(&(m_streamWithhigh[0]),hipStreamDefault, priority_high);

  for (int i = 0; i < stream_num; i++)
    hipStreamCreateWithPriority(&(m_streamWithlow[i]),hipStreamDefault, priority_low);

  notifyGpuMgr = notify_callback;
  #if 0
  // For debug
  printf("Size of hipStream_t is %ld\n", sizeof(hipStream_t));
  # endif
}

void GPU_Deinit(void)
{
  hipDeviceReset();
}

void CUDART_CB launchFuncCallback(void *data){
  pid_t pid = getpid();
  printf("launchFuncCallback %lu: Run launchFuncCallback!\n",pid);
  HostFuncData tran_data = *((HostFuncData *)data);

  // Free the memory space
  free(data);

  // Unique entry to notify GPU Manager GPU operation done on stream_idx
  // 通知gpu mgr gpu opt已完成 解锁阻塞的线程
  //只有同步方式需要执行
  if(tran_data.IsNotify){
    //sleep一会，以保证gpu_mgr 已经处于wait
    usleep(3000);//3ms
    notifyGpuMgr(tran_data.gopt_cv_mutex,tran_data.gopt_cv);
  }
  else{//只有异步方式执行
    //run post Func
    (*(tran_data.Post_Function))();
    //执行到此处时，表示一个GPU_Operation全部被执行完毕
    //delete new的GPU_Operation对象
    printf("launchFuncCallback: run delete gpu opt!\n");
    delete tran_data.gpu_opt;
  }
  
  printf("launchFuncCallback %lu: launchFuncCallback end!\n",pid);
}

cuStreamPtr_t getCuStreamPtr(int stream_idx)
{
  // TODO: Range Check
  return (void *)&(m_stream[stream_idx]);
}

cuStreamPtr_t getCuLowStreamPtr(int stream_idx)
{
  // TODO: Range Checkm_stream
  return (void *)&(m_streamWithlow[stream_idx]);
}

cuStreamPtr_t getCuHighStreamPtr()
{
  // TODO: Range Checkm_stream
  return (void *)&(m_streamWithhigh[0]);
}

void LaunchHostFunc(bool IsSync,
                    GPU_Post_Function *post_func,
                    std::mutex *gopt_CvMetux,
                    std::condition_variable *gopt_Cv,
                    void *stream_ptr,
                    void *g_op){

  // printf("LaunchHostFunc: run launch Host Func!\n");
  //准备需要传递给launchFuncCallback函数的数据
  HostFuncData *hostfuncdata = (HostFuncData *)malloc(sizeof(HostFuncData));
  hostfuncdata->IsNotify = IsSync;
  hostfuncdata->Post_Function = post_func;
  hostfuncdata->gopt_cv_mutex = gopt_CvMetux;
  hostfuncdata->gopt_cv = gopt_Cv;
  hostfuncdata->gpu_opt = g_op;
  hipStream_t *s_ptr;
  hipError_t error;
  // printf("LaunchHostFunc: run hipLaunchHostFunc!\n");
  if(stream_ptr == NULL){
    hostfuncdata->gopt_stm = NULL;
    error = hipLaunchHostFunc(0, launchFuncCallback, hostfuncdata);//default stream
    if (error != hipSuccess){
      printf("LaunchHostFunc: hipLaunchHostFunc Error:%s\n", GetRuntimeError(error));
    }
  }
  else{
    s_ptr = (hipStream_t *)stream_ptr;
    hostfuncdata->gopt_stm = s_ptr;
    error = hipLaunchHostFunc(*s_ptr, launchFuncCallback, hostfuncdata);
    if (error != hipSuccess){
      printf("LaunchHostFunc: hipLaunchHostFunc Error:%s\n", GetRuntimeError(error));
    }
  }
  // printf("LaunchHostFunc: launch Host Func end!\n");
}

const char *GetRuntimeError(hipError_t error)
{
    if (error != hipSuccess)
    {
        return hipGetErrorString(error);
    }
    else
        return NULL;
}


#endif