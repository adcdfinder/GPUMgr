#include "hip/hip_runtime.h"
#include "../../inc/mini_type.hpp"
#include "util.cu"
#include "../../inc/gpu_mgr/gpu_mgr.cuh"
#include <stdio.h>
#include <sstream>
#include <string>
// #define TEST_DEBUG
#define MINI_N 10


uint32_t *h_sum;
uint32_t *d_sum;
int m = 1;
float occupy = 0.0;
//ROS2 需要gpu执行的操作
__device__ volatile int g_stopFlag = 0;
int stopFlag = 0;

__global__ void noiseKernel(uint32_t smID){
  if(smID == getSMID()){
    printf("exit on %d .\n", smID);
    asm("exit;");
  }
  while(!g_stopFlag){
     MySleep(1); // sleep for 1 millisecond
  }

}

void syncNoiseKernel(){
  hipError_t cudaStatus = hipMemcpyToSymbol(HIP_SYMBOL(g_stopFlag), &stopFlag, 0, hipMemcpyHostToDevice);

}

void resetNoiseFlag(){
  stopFlag = 0;
  syncNoiseKernel();
  hipDeviceSynchronize();
}

void stopNoiseFlag(){
  stopFlag = 1;
  syncNoiseKernel();
  hipDeviceSynchronize();
  printf("success ");
}

__global__ void kAddXandY0(uint32_t *sum, uint32_t x, uint32_t y)
{
  sum[0] = x + y;
  // busySleep(50000000); // busy sleep about 40ms
  // MySleep(500);//sleep 500ms
  return;
}

__global__ void kAddXandY1(uint32_t *sum, uint32_t x, uint32_t y)
{
  // MySleep(50000);
  for(int i  = 0; i < 500;i++){
    continue;
  }
  // printf("Current task is running on %d sm .", getSMID());
  // busySleep(50000000); // busy sleep about 40ms
  // MySleep(500);//sleep 500ms
  return;
}

// __global__ void normalTask(uint32_t *block, uint32_t* timestamp, uint32_t *smId){
//   int index = blockIdx.x * blockDim.x + threadIdx.x;
//   block[index] = blockIdx.x;
//   timestamp[index] = clock();
//   smId[index] = getSMID();

// }

__global__ void testKernel()
{
  // MySleep(50000);
  int start = clock();
  while(clock() - start < 100000){
    
  }
  // for(int i  = 0; i < 5000; i++){
  //   continue;
  // }
  // // printf("Current task is running on %d sm .", getSMID());
  // busySleep(50000000); // busy sleep about 40ms
  // MySleep(500);//sleep 500ms
  return;


}

__global__ void testAffinityKernel()
{
  if(blockIdx.x == (gridDim.x - 1)){
    g_stopFlag = 1;
  }

  printf("Affinity task is running on %d sm .\n", getSMID());

  int start =clock();
  while(clock() - start < 100000){
   
  }
  // for(int i  = 0; i < 5000; i++){
  //   continue;
  // }
}

void mini_AddXandY0(void *ka_ptr, void *stream)
{
  hipStream_t *stream_ptr = (hipStream_t *)stream;
  uint32_t x = ((mini_t *)ka_ptr)->x;
  uint32_t y = ((mini_t *)ka_ptr)->y;

  kAddXandY0<<<1, 1, 0, *stream_ptr>>>(d_sum, x, y);

  cuMemCpyAsyncDtoH(h_sum, d_sum,sizeof(uint32_t)*MINI_N,*stream_ptr);
  // printf("mini_AddXandY0: gpu mgr AsyncDtoH end!\n");
}

void mini_AddXandY1(void *ka_ptr, void *stream, int blocksize = 1 , int numblocks = 1)
{
  // hipStream_t *stream_ptr = (hipStream_t *)stream;


  // printf("testkernel is executing. /n");

  testKernel<<< numblocks, blocksize, 0 ,*((hipStream_t *)stream)>>>();
  // hipDeviceSynchronize();
  // *d_sum = 10;
  return;


  // uint32_t x = ((mini_t *)ka_ptr)->x;
  // uint32_t y = ((mini_t *)ka_ptr)->y;

  // kAddXandY1<<<blocksize, numblocks, 0, *stream_ptr>>>(d_sum, x, y);

  // cuMemCpyAsyncDtoH(h_sum, d_sum,sizeof(uint32_t)*MINI_N,*stream_ptr);
  // // printf("mini_AddXandY1: gpu mgr AsyncDtoH end!\n");
}

void mini_AddXandY1_Affinity(void *ka_ptr, void *stream, int blocksize = 1 , int numblocks = 1)
{
  // hipStream_t *stream_ptr = (hipStream_t *)stream;

  printf("testkernel is executing. /n");

  testAffinityKernel<<< numblocks, blocksize,  0 ,*((hipStream_t *)stream)>>>();
  // hipDeviceSynchronize();
  // *d_sum = 10;
  return;
  
}

void startNoiseKernel(int blocksize, int numblocks, void *stream){
  hipStream_t *stream_ptr = (hipStream_t *)stream;
  noiseKernel<<<numblocks, blocksize,  0, (*(hipStream_t *)stream_ptr)>>>(5);
}

int mini_GetResult(void)
{
  return *h_sum;
}

const char *MyGetRuntimeError(hipError_t error)
{
    if (error != hipSuccess)
    {
        return hipGetErrorString(error);
    }
    else
        return NULL;
}

void mini_PostFunc0()
{
  //此处会报CUDA错误：CUDA API不允许在host func中执行
  // hipError_t error;
  // printf("PostFunc run DtoH!\n");
  // error = cuMemCpyAsyncDtoH(h_sum, d_sum,sizeof(uint32_t)*MINI_N,stm);
  // if (error != hipSuccess){
  //    printf("thread hipOccupancyMaxActiveBlocksPerMultiprocessor Error:%s\n", MyGetRuntimeError(error));
  // }
  printf("h_sum is ");
  for (int i = 0; i < MINI_N; i++)
  {
    printf("%d ",h_sum[i]);
  }

  printf("\n");
}

void mini_PostFunc1()
{
  //此处会报CUDA错误：CUDA API不允许在host func中执行
  // hipError_t error;
  // printf("PostFunc run DtoH!\n");
  // error = cuMemCpyAsyncDtoH(h_sum, d_sum,sizeof(uint32_t)*MINI_N,stm);
  // if (error != hipSuccess){
  //    printf("thread hipOccupancyMaxActiveBlocksPerMultiprocessor Error:%s\n", MyGetRuntimeError(error));
  // }
  printf("h_sum is ");
  for (int i = 0; i < MINI_N; i++)
  {
    printf("%d ",h_sum[i]);
  }

  printf("\n");
}

void mini_Init(void)
{

  // Allocate host memory
  hipHostMalloc((void **)&h_sum, sizeof(uint32_t) * MINI_N);

  // Allocate GPU memory
  hipMalloc((void **)&d_sum, sizeof(uint32_t) * MINI_N);

  // Initialize var on host and GPU memory
  h_sum[9] = 666;
  hipMemcpy(d_sum, h_sum, sizeof(uint32_t) *MINI_N, hipMemcpyHostToDevice); // copy to gpu
}

void mini_Deinit(void){
  hipHostFree(h_sum);
}

float get_gpu_utilization() {
    FILE* pipe = popen("nvidia-smi --query-gpu=utilization.gpu --format=csv,noheader,nounits", "r");
    if (!pipe) {
        throw std::runtime_error("popen() failed!");
    }

    char buffer[128];
    float gpu_utilization = -1.0f;

    if (fgets(buffer, 128, pipe) != nullptr) {
        // Read the output and parse GPU utilization
        std::istringstream ss(buffer);
        std::string token;
        std::getline(ss, token, ',');  // Ignore the index
        std::getline(ss, token, ',');  // Get utilization
        gpu_utilization = std::stof(token);
    }

    pclose(pipe);
    return gpu_utilization;
}

void launchNoiseTest(hipDeviceProp_t prop, void *noiseStream, void *workStream, bool bPolicyApplied , float * elapse){
  // int m = 9;
  int smNum = prop.multiProcessorCount;
  int threadsPerBlock = (prop.maxThreadsPerBlock - 256)/m;
  float currentOcc = 0.0;

  //for record the elapse
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  hipEventQuery(start);
  // printf("Executing kernel");

  // Kernerl operations
  startNoiseKernel(threadsPerBlock, smNum * m , noiseStream);
  mini_AddXandY1_Affinity(nullptr, workStream, prop.maxThreadsPerBlock, 16);
  if(!bPolicyApplied){
    hipDeviceSynchronize();
  }
  mini_AddXandY1(nullptr, workStream, 256, 1);
  if(!bPolicyApplied){

    hipDeviceSynchronize();
  }

  if(bPolicyApplied){
    currentOcc = get_gpu_utilization();
  }
  

  mini_AddXandY1(nullptr, workStream, 256, 1);
  
  hipDeviceSynchronize();
  resetNoiseFlag();


  //for record the elapse

  hipEventRecord(stop);
  hipEventSynchronize(stop);
  hipEventElapsedTime(elapse, start, stop);

  hipEventDestroy(start);
  hipEventDestroy(stop);

  *elapse = occupy;

}


#ifdef TEST_DEBUG
int main(){
  
  hipStream_t *m_stream = (hipStream_t *)malloc(  sizeof(hipStream_t *));
  hipStream_t *m_stream_noise = (hipStream_t *)malloc(  sizeof(hipStream_t *));
  int priority_high, priority_low;

  hipDeviceProp_t prop;
  int device;
  hipGetDevice(&device);
  hipGetDeviceProperties(&prop, 0);

  
  hipDeviceGetStreamPriorityRange(&priority_low,&priority_high);
  // create streams with highest and lowest available priorities
  // hipStream_t st_high, st_low;

  hipStreamCreateWithPriority(&(m_stream[0]),hipStreamDefault, priority_low);
  hipStreamCreateWithPriority(&(m_stream_noise[0]),hipStreamDefault, priority_high);

  printf("the highest priority is %d . \n", priority_high);
  printf("the lowest priority is %d . \n", priority_low);
  
  int smNum = prop.multiProcessorCount;
  int threadsPerBlock = prop.maxThreadsPerBlock - 500;
  int threadsPerSM = prop.maxThreadsPerMultiProcessor;
  printf("Compute capability: %d.%d\n", prop.major, prop.minor);
  printf("SM num: %d.\n", smNum);

  // return 0;
  printf("the max threads perblock is %d . \n", threadsPerBlock);
  printf("the max threads per sm is %d . \n", threadsPerSM);
  // startNoiseKernel(threadsPerBlock, smNum, m_stream_noise );

  // mini_AddXandY1_Affinity(nullptr, m_stream , 512, 8);
  mini_AddXandY1(nullptr, m_stream , 86, 14);
  // mini_AddXandY1(nullptr, m_stream , 256, 1);
  
  // hipDeviceSynchronize();
  resetNoiseFlag();


  return 0;


}
#endif